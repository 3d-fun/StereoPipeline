// VW
//#include <vw/vw.h>

// includes, system
//#include <iostream>

// includes, project
#include <hip/hip_runtime.h>

/* --------------------------- target code ------------------------------*/  
#define ACOS_THREAD_CNT 128
#define N 128

struct acosParams { 
    float *arg; 
    float *res; 
    int n; 
}; 
 
__global__ void acos_main (struct acosParams parms) 
{ 
    int i; 
    for (i = threadIdx.x; i < parms.n; i += ACOS_THREAD_CNT) { 
        parms.res[i] = acosf(parms.arg[i]); 
    } 
} 
 
/* --------------------------- host code ------------------------------*/ 
 
int main (int argc, char *argv[]) 
{ 
    hipError_t         cudaStat; 
    float*              acosRes = 0; 
    float*              acosArg = 0; 
    float* arg[N];
    float* res[N];
    struct acosParams   funcParams; 
 
    //    ... fill arguments array ‘arg’ .... 
 
    //cout << "Hello, world3.\n";
    cudaStat = hipMalloc ((void **)&acosArg, N * sizeof(acosArg[0])); 
    cudaStat = hipMemcpy (acosArg, arg, N * sizeof(arg[0]), hipMemcpyHostToDevice); 
 
    funcParams.res = acosRes; 
    funcParams.arg = acosArg; 
    funcParams.n = N; 
     
    acos_main<<<1,ACOS_THREAD_CNT>>>(funcParams); 

    cudaStat = hipMemcpy (res, acosRes, N * sizeof(res[0]), hipMemcpyDeviceToHost); 
 
    //    ... process result array ‘res’ .... 
} 

