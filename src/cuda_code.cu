#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

/* --------------------------- target code ------------------------------*/  
#define THREAD_COUNT 128

struct params { 
    float *input; 
    float *output; 
    int n; 
}; 
 
__global__ void flip_main (struct params p) 
{ 
  int i; 
  for (i = threadIdx.x; i < p.n; i += THREAD_COUNT) { 
    p.output[i] = 1.0f - p.input[i]; 
  } 
} 
 
/* --------------------------- host code ------------------------------*/ 

// Invert a 1 channel, 32-bit floating point image.
void invert_image (float* img, int width, int height) { 
    hipError_t         cudaStat; 
    float*              out = 0; 
    float*              in = 0; 
    struct params   funcParams; 
    int N = width * height;
 
    cout << "Allocating GPU Memory.\n";
    cudaStat = hipMalloc ((void **)&in, N * sizeof(in[0])); 
    cudaStat = hipMalloc ((void **)&out, N * sizeof(out[0])); 

    cout << "Copying data.\n";
    cudaStat = hipMemcpy (in, img, N * sizeof(img[0]), hipMemcpyHostToDevice); 
 
    funcParams.output = out; 
    funcParams.input = in; 
    funcParams.n = N; 
     
    cout << "Running kernel.\n";
    flip_main<<<1,THREAD_COUNT>>>(funcParams); 

    cout << "Copying result.\n";
    cudaStat = hipMemcpy (img, out, N * sizeof(out[0]), hipMemcpyDeviceToHost); 
} 

